#include "hip/hip_runtime.h"
#include "dedisperse.h"
#include "dedisperse_kernel.cuh"

#include "CU.h"


// Kernel tuning parameters
#define DEDISP_BLOCK_SIZE       256
#define DEDISP_BLOCK_SAMPS      8

/*
 * Helper functions
 */
void DedispKernel::copy_delay_table(
    const void* src,
    size_t count,
    size_t offset,
    hipStream_t stream)
{
    cu::checkError(hipMemcpyToSymbolAsync(HIP_SYMBOL(
        c_delay_table),
        src,
        count, offset,
        hipMemcpyDeviceToDevice, stream)
    );
}

void DedispKernel::copy_killmask(
    const void* src,
    size_t count,
    size_t offset,
    hipStream_t stream)
{
    cu::checkError(hipMemcpyToSymbolAsync(HIP_SYMBOL(
        c_killmask),
        src,
        count, offset,
        hipMemcpyDeviceToDevice, stream)
    );
}

unsigned int DedispKernel::get_nsamps_per_thread()
{
    return DEDISP_SAMPS_PER_THREAD;
}

/*
 * dedisperse routine
 */
void DedispKernel::launch(
    const dedisp_word*  d_in,
    dedisp_size         in_stride,
    dedisp_size         nsamps,
    dedisp_size         in_nbits,
    dedisp_size         nchans,
    dedisp_size         chan_stride,
    const dedisp_float* d_dm_list,
    dedisp_size         dm_count,
    dedisp_size         dm_stride,
    dedisp_byte*        d_out,
    dedisp_size         out_stride,
    dedisp_size         out_nbits,
    hipStream_t        stream)
{
    enum {
        BITS_PER_BYTE            = 8,
        BYTES_PER_WORD           = sizeof(dedisp_word) / sizeof(dedisp_byte),
        BLOCK_DIM_X              = DEDISP_BLOCK_SAMPS,
        BLOCK_DIM_Y              = DEDISP_BLOCK_SIZE / DEDISP_BLOCK_SAMPS,
        MAX_CUDA_GRID_SIZE_X     = 65535,
        MAX_CUDA_GRID_SIZE_Y     = 65535,
        MAX_CUDA_1D_TEXTURE_SIZE = (1<<27)
    };

    // Initialise texture memory if necessary
    // --------------------------------------
    if (m_use_texture_mem && d_in != m_d_in)
    {
        m_d_in = (dedisp_word *) d_in;

        dedisp_size chans_per_word = sizeof(dedisp_word)*BITS_PER_BYTE / in_nbits;
        dedisp_size nchan_words    = nchans / chans_per_word;
        dedisp_size input_words    = in_stride * nchan_words;

        // Check the texture size limit
        if( input_words > MAX_CUDA_1D_TEXTURE_SIZE ) {
            throw_error(DEDISP_INTERNAL_GPU_ERROR);
        }

        // Bind the texture memory
        hipChannelFormatDesc channel_desc = hipCreateChannelDesc<dedisp_word>();
        cu::checkError(hipBindTexture(
            0, t_in, d_in, channel_desc,
            input_words * sizeof(dedisp_word))
        );
    }
    // --------------------------------------

    // Define thread decomposition
    // Note: Block dimensions x and y represent time samples and DMs respectively
    dim3 block(BLOCK_DIM_X,
               BLOCK_DIM_Y);
    // Note: Grid dimension x represents time samples. Dimension y represents DMs

    // Divide and round up
    dedisp_size nsamp_blocks = (nsamps - 1)
        / ((dedisp_size)DEDISP_SAMPS_PER_THREAD*block.x) + 1;
    dedisp_size ndm_blocks   = (dm_count - 1) / (dedisp_size)block.y + 1;

    // Constrain the grid size to the maximum allowed
    ndm_blocks = min((unsigned int)ndm_blocks,
                     (unsigned int)(MAX_CUDA_GRID_SIZE_Y));

    dim3 grid(nsamp_blocks, ndm_blocks);

    // Divide and round up
    dedisp_size nsamps_reduced = (nsamps - 1) / DEDISP_SAMPS_PER_THREAD + 1;

    // Execute the kernel
#define DEDISP_CALL_KERNEL(NBITS, USE_TEXTURE_MEM)						\
    dedisperse_kernel<NBITS,DEDISP_SAMPS_PER_THREAD,BLOCK_DIM_X,        \
                      BLOCK_DIM_Y,USE_TEXTURE_MEM>                      \
        <<<grid, block, 0, stream>>>(d_in,								\
                                     nsamps,							\
                                     nsamps_reduced,					\
                                     nsamp_blocks,						\
                                     in_stride,							\
                                     dm_count,							\
                                     dm_stride,							\
                                     ndm_blocks,						\
                                     nchans,							\
                                     chan_stride,						\
                                     d_out,								\
                                     out_nbits,							\
                                     out_stride,						\
                                     d_dm_list)
    // Note: Here we dispatch dynamically on nbits for supported values
    if( m_use_texture_mem ) {
        switch( in_nbits ) {
            case 1:  DEDISP_CALL_KERNEL(1,true);  break;
            case 2:  DEDISP_CALL_KERNEL(2,true);  break;
            case 4:  DEDISP_CALL_KERNEL(4,true);  break;
            case 8:  DEDISP_CALL_KERNEL(8,true);  break;
            case 16: DEDISP_CALL_KERNEL(16,true); break;
            case 32: DEDISP_CALL_KERNEL(32,true); break;
            default: /* should never be reached */ break;
        }
    }
    else {
        switch( in_nbits ) {
            case 1:  DEDISP_CALL_KERNEL(1,false);  break;
            case 2:  DEDISP_CALL_KERNEL(2,false);  break;
            case 4:  DEDISP_CALL_KERNEL(4,false);  break;
            case 8:  DEDISP_CALL_KERNEL(8,false);  break;
            case 16: DEDISP_CALL_KERNEL(16,false); break;
            case 32: DEDISP_CALL_KERNEL(32,false); break;
            default: /* should never be reached */ break;
        }
    }
#undef DEDISP_CALL_KERNEL
}