#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "DedispPlan.hpp"
#include "transpose/transpose.hpp"
#include "gpu_memory.hpp"

#include "dedisp_defines.h"
#include "dedisp_error.hpp"
#include "dedisp_kernels.hpp"
#include "dedisperse/dedisperse.h"
#include "unpack/unpack.h"

#if defined(DEDISP_BENCHMARK)
#include <iostream>
#include <fstream>
using std::cout;
using std::endl;
#include "external/Stopwatch.h"
#endif

namespace dedisp
{

// Constructor
DedispPlan::DedispPlan(size_type  nchans,
                       float_type dt,
                       float_type f0,
                       float_type df) :
    d_dm_list(0)
{
    if( hipGetLastError() != hipSuccess ) {
        throw_error(DEDISP_PRIOR_GPU_ERROR);
    }

    int device_idx;
    hipGetDevice(&device_idx);

    // Check for parameter errors
    if( nchans > DEDISP_MAX_NCHANS ) {
        throw_error(DEDISP_NCHANS_EXCEEDS_LIMIT);
    }

    // Force the df parameter to be negative such that
    //   freq[chan] = f0 + chan * df.
    df = -abs(df);

    m_dm_count      = 0;
    m_nchans        = nchans;
    m_gulp_size     = DEDISP_DEFAULT_GULP_SIZE;
    m_max_delay     = 0;
    m_dt            = dt;
    m_f0            = f0;
    m_df            = df;
    //m_stream        = 0;

    // Generate delay table and copy to device memory
    // Note: The DM factor is left out and applied during dedispersion
    h_delay_table.resize(nchans);
    kernel::generate_delay_table(h_delay_table.data(), nchans, dt, f0, df);
    try {
        d_delay_table.resize(nchans);
    }
    catch(...) {
        throw_error(DEDISP_MEM_ALLOC_FAILED);
    }
    try {
        d_delay_table = h_delay_table;
    }
    catch(...) {
        throw_error(DEDISP_MEM_COPY_FAILED);
    }

    // Initialise the killmask
    h_killmask.resize(nchans, (dedisp_bool)true);
    try {
        d_killmask.resize(nchans);
    }
    catch(...) {
        throw_error(DEDISP_MEM_ALLOC_FAILED);
    }
    set_killmask((dedisp_bool*)0);
}

// Destructor
DedispPlan::~DedispPlan() {
}

// Private helper functions
template<typename T>
T min(T a, T b) { return a<b ? a : b; }
unsigned long div_round_up(unsigned long a, unsigned long b) {
    return (a-1) / b + 1;
}

// Public interface
void DedispPlan::set_device(int device_idx) {
    if( hipGetLastError() != hipSuccess ) {
        throw_error(DEDISP_PRIOR_GPU_ERROR);
    }

    hipError_t error = hipSetDevice(device_idx);
    // Note: hipErrorInvalidValue isn't a documented return value, but
    //         it still gets returned :/
    if( hipErrorInvalidDevice == error ||
        hipErrorInvalidValue == error )
        throw_error(DEDISP_INVALID_DEVICE_INDEX);
    else if( hipErrorSetOnActiveProcess == error )
        throw_error(DEDISP_DEVICE_ALREADY_SET);
    else if( hipSuccess != error )
        throw_error(DEDISP_UNKNOWN_ERROR);
}

void DedispPlan::set_gulp_size(size_type gulp_size) {
    m_gulp_size = gulp_size;
}

void DedispPlan::set_killmask(const bool_type* killmask)
{
    if( hipGetLastError() != hipSuccess ) {
        throw_error(DEDISP_PRIOR_GPU_ERROR);
    }
    if( 0 != killmask ) {
        // Copy killmask to plan (both host and device)
        h_killmask.assign(killmask, killmask + m_nchans);
        try {
            d_killmask = h_killmask;
        }
        catch(...) { throw_error(DEDISP_MEM_COPY_FAILED); }
    }
    else {
        // Set the killmask to all true
        std::fill(h_killmask.begin(), h_killmask.end(), (dedisp_bool)true);
        thrust::fill(d_killmask.begin(), d_killmask.end(),
                     (dedisp_bool)true);
    }
}

void DedispPlan::set_dm_list(const float_type* dm_list,
                             size_type         count)
{
    if( !dm_list ) {
        throw_error(DEDISP_INVALID_POINTER);
    }
    if( hipGetLastError() != hipSuccess ) {
        throw_error(DEDISP_PRIOR_GPU_ERROR);
    }

    m_dm_count = count;
    h_dm_list.assign(dm_list, dm_list+count);

    // Copy to the device
    try {
        d_dm_list.resize(m_dm_count);
    }
    catch(...) { throw_error(DEDISP_MEM_ALLOC_FAILED); }
    try {
        d_dm_list = h_dm_list;
    }
    catch(...) { throw_error(DEDISP_MEM_COPY_FAILED); }

    // Calculate the maximum delay and store it in the plan
    m_max_delay = dedisp_size(h_dm_list[m_dm_count-1] *
                              h_delay_table[m_nchans-1] + 0.5);
}

void DedispPlan::generate_dm_list(float_type dm_start,
                                  float_type dm_end,
                                  float_type ti,
                                  float_type tol)
{
    if( hipGetLastError() != hipSuccess ) {
        throw_error(DEDISP_PRIOR_GPU_ERROR);
    }

    // Generate the DM list (on the host)
    h_dm_list.clear();
    kernel::generate_dm_list(
        h_dm_list,
        dm_start, dm_end,
        m_dt, ti, m_f0, m_df,
        m_nchans, tol);
    m_dm_count = h_dm_list.size();

    // Allocate device memory for the DM list
    try {
        d_dm_list.resize(m_dm_count);
    }
    catch(...) { throw_error(DEDISP_MEM_ALLOC_FAILED); }
    try {
        d_dm_list = h_dm_list;
    }
    catch(...) { throw_error(DEDISP_MEM_COPY_FAILED); }

    // Calculate the maximum delay and store it in the plan
    m_max_delay = dedisp_size(h_dm_list[m_dm_count-1] *
                              h_delay_table[m_nchans-1] + 0.5);
}

void DedispPlan::execute(size_type        nsamps,
                         const byte_type* in,
                         size_type        in_nbits,
                         byte_type*       out,
                         size_type        out_nbits,
                         unsigned         flags)
{
    enum {
        BITS_PER_BYTE = 8
    };

    // Note: The default out_stride is nsamps - m_max_delay
    dedisp_size out_bytes_per_sample =
        out_nbits / (sizeof(dedisp_byte) * BITS_PER_BYTE);

    // Note: Must be careful with integer division
    dedisp_size in_stride =
        m_nchans * in_nbits / (sizeof(dedisp_byte) * BITS_PER_BYTE);
    dedisp_size out_stride = (nsamps - m_max_delay) * out_bytes_per_sample;

    execute_adv(
        nsamps,
        in, in_nbits, in_stride,
        out, out_nbits, out_stride,
        flags);
}

void DedispPlan::execute_adv(size_type        nsamps,
                             const byte_type* in,
                             size_type        in_nbits,
                             size_type        in_stride,
                             byte_type*       out,
                             size_type        out_nbits,
                             size_type        out_stride,
                             unsigned         flags)
{
    dedisp_size first_dm_idx = 0;
    dedisp_size dm_count = m_dm_count;

    execute_guru(
        nsamps,
        in, in_nbits, in_stride,
        out, out_nbits, out_stride,
        first_dm_idx, dm_count,
        flags);
}

void DedispPlan::execute_guru(size_type        nsamps,
                              const byte_type* in,
                              size_type        in_nbits,
                              size_type        in_stride,
                              byte_type*       out,
                              size_type        out_nbits,
                              size_type        out_stride,
                              size_type        first_dm_idx,
                              size_type        dm_count,
                              unsigned         flags)
{
    if( hipGetLastError() != hipSuccess ) {
        throw_error(DEDISP_PRIOR_GPU_ERROR);
    }

    enum {
        BITS_PER_BYTE  = 8,
        BYTES_PER_WORD = sizeof(dedisp_word) / sizeof(dedisp_byte)
    };

    dedisp_size out_bytes_per_sample = out_nbits / (sizeof(dedisp_byte) *
                                                    BITS_PER_BYTE);

    if( 0 == in || 0 == out ) {
        throw_error(DEDISP_INVALID_POINTER);
    }
    // Note: Must be careful with integer division
    if( in_stride < m_nchans*in_nbits/(sizeof(dedisp_byte)*BITS_PER_BYTE) ||
        out_stride < (nsamps - m_max_delay)*out_bytes_per_sample ) {
        throw_error(DEDISP_INVALID_STRIDE);
    }
    if( 0 == m_dm_count ) {
        throw_error(DEDISP_NO_DM_LIST_SET);
    }
    if( nsamps < m_max_delay ) {
        throw_error(DEDISP_TOO_FEW_NSAMPS);
    }

    // Check for valid synchronisation flags
    if( flags & DEDISP_ASYNC && flags & DEDISP_WAIT ) {
        throw_error(DEDISP_INVALID_FLAG_COMBINATION);
    }

    // Check for valid nbits values
    if( in_nbits  != 1 &&
        in_nbits  != 2 &&
        in_nbits  != 4 &&
        in_nbits  != 8 &&
        in_nbits  != 16 &&
        in_nbits  != 32 ) {
        throw_error(DEDISP_UNSUPPORTED_IN_NBITS);
    }
    if( out_nbits != 8 &&
        out_nbits != 16 &&
        out_nbits != 32 ) {
        throw_error(DEDISP_UNSUPPORTED_OUT_NBITS);
    }

    bool using_host_memory = false;
    if( flags & DEDISP_HOST_POINTERS && flags & DEDISP_DEVICE_POINTERS ) {
        throw_error(DEDISP_INVALID_FLAG_COMBINATION);
    }
    else {
        using_host_memory = !(flags & DEDISP_DEVICE_POINTERS);
    }

    // Copy the lookup tables to constant memory on the device
    copy_delay_table(thrust::raw_pointer_cast(&d_delay_table[0]),
                     m_nchans * sizeof(dedisp_float),
                     0, 0);
    copy_killmask(thrust::raw_pointer_cast(&d_killmask[0]),
                  m_nchans * sizeof(dedisp_bool),
                  0, 0);

    // Compute the problem decomposition
    dedisp_size nsamps_computed = nsamps - m_max_delay;
    // Specify the maximum gulp size
    dedisp_size nsamps_computed_gulp_max;
    if( using_host_memory ) {
        nsamps_computed_gulp_max = min(m_gulp_size, nsamps_computed);
    }
    else {
        // Just do it in one gulp if given device pointers
        nsamps_computed_gulp_max = nsamps_computed;
    }

    // Just to be sure
    // TODO: This seems quite wrong. Why was it here?
    /*
    if( nsamps_computed_gulp_max < m_max_delay ) {
        throw_error(DEDISP_TOO_FEW_NSAMPS);
    }
    */

    // Compute derived counts for maximum gulp size [dedisp_word == 4 bytes]
    dedisp_size nsamps_gulp_max = nsamps_computed_gulp_max + m_max_delay;
    dedisp_size chans_per_word  = sizeof(dedisp_word)*BITS_PER_BYTE / in_nbits;
    dedisp_size nchan_words     = m_nchans / chans_per_word;

    // We use words for processing but allow arbitrary byte strides, which are
    //   not necessarily friendly.
    bool friendly_in_stride = (0 == in_stride % BYTES_PER_WORD);

    // Note: If desired, this could be rounded up, e.g., to a power of 2
    dedisp_size in_buf_stride_words      = nchan_words;
    dedisp_size in_count_gulp_max        = nsamps_gulp_max * in_buf_stride_words;

    dedisp_size nsamps_padded_gulp_max   = div_round_up(nsamps_computed_gulp_max,
                                                        DEDISP_SAMPS_PER_THREAD)
        * DEDISP_SAMPS_PER_THREAD + m_max_delay;
    dedisp_size in_count_padded_gulp_max =
        nsamps_padded_gulp_max * in_buf_stride_words;

    // TODO: Make this a parameter?
    dedisp_size min_in_nbits = 0;
    dedisp_size unpacked_in_nbits = max((int)in_nbits, (int)min_in_nbits);
    dedisp_size unpacked_chans_per_word =
        sizeof(dedisp_word)*BITS_PER_BYTE / unpacked_in_nbits;
    dedisp_size unpacked_nchan_words = m_nchans / unpacked_chans_per_word;
    dedisp_size unpacked_buf_stride_words = unpacked_nchan_words;
    dedisp_size unpacked_count_padded_gulp_max =
        nsamps_padded_gulp_max * unpacked_buf_stride_words;

    dedisp_size out_stride_gulp_samples  = nsamps_computed_gulp_max;
    dedisp_size out_stride_gulp_bytes    =
        out_stride_gulp_samples * out_bytes_per_sample;
    dedisp_size out_count_gulp_max       = out_stride_gulp_bytes * dm_count;

    // Organise device memory pointers
    // -------------------------------
    const dedisp_word* d_in = 0;
    dedisp_word*       d_transposed = 0;
    dedisp_word*       d_unpacked = 0;
    dedisp_byte*       d_out = 0;
    thrust::device_vector<dedisp_word> d_in_buf;
    thrust::device_vector<dedisp_word> d_transposed_buf;
    thrust::device_vector<dedisp_word> d_unpacked_buf;
    thrust::device_vector<dedisp_byte> d_out_buf;
    // Allocate temporary buffers on the device where necessary
    if( using_host_memory || !friendly_in_stride ) {
        try { d_in_buf.resize(in_count_gulp_max); }
        catch(...) { throw_error(DEDISP_MEM_ALLOC_FAILED); }
        d_in = thrust::raw_pointer_cast(&d_in_buf[0]);
    }
    else {
        d_in = (dedisp_word*)in;
    }
    if( using_host_memory ) {
        try { d_out_buf.resize(out_count_gulp_max); }
        catch(...) { throw_error(DEDISP_MEM_ALLOC_FAILED); }
        d_out = thrust::raw_pointer_cast(&d_out_buf[0]);
    }
    else {
        d_out = out;
    }
    try { d_transposed_buf.resize(in_count_padded_gulp_max); }
    catch(...) { throw_error(DEDISP_MEM_ALLOC_FAILED); }
    d_transposed = thrust::raw_pointer_cast(&d_transposed_buf[0]);

    try { d_unpacked_buf.resize(unpacked_count_padded_gulp_max); }
    catch(...) { throw_error(DEDISP_MEM_ALLOC_FAILED); }
    d_unpacked = thrust::raw_pointer_cast(&d_unpacked_buf[0]);
    // -------------------------------

    // TODO: Eventually re-implement streams
    hipStream_t stream = 0;//(hipStream_t)m_stream;

#ifdef DEDISP_BENCHMARK
    std::unique_ptr<Stopwatch> copy_to_timer(Stopwatch::create());
    std::unique_ptr<Stopwatch> copy_from_timer(Stopwatch::create());
    std::unique_ptr<Stopwatch> transpose_timer(Stopwatch::create());
    std::unique_ptr<Stopwatch> kernel_timer(Stopwatch::create());
#endif

    // Gulp loop
    for( dedisp_size gulp_samp_idx=0;
         gulp_samp_idx<nsamps_computed;
         gulp_samp_idx+=nsamps_computed_gulp_max ) {

        dedisp_size nsamps_computed_gulp = min(nsamps_computed_gulp_max,
                                               nsamps_computed-gulp_samp_idx);
        dedisp_size nsamps_gulp          = nsamps_computed_gulp + m_max_delay;
        dedisp_size nsamps_padded_gulp   = div_round_up(nsamps_computed_gulp,
                                                        DEDISP_SAMPS_PER_THREAD)
            * DEDISP_SAMPS_PER_THREAD + m_max_delay;

#ifdef DEDISP_BENCHMARK
        copy_to_timer->Start();
#endif
        // Copy the input data from host to device if necessary
        if( using_host_memory ) {
            // Allowing arbitrary byte strides means we must do a strided copy
            if( !copy_host_to_device_2d((dedisp_byte*)d_in,
                                        in_buf_stride_words * BYTES_PER_WORD,
                                        in + gulp_samp_idx*in_stride,
                                        in_stride,
                                        nchan_words * BYTES_PER_WORD,
                                        nsamps_gulp) ) {
                throw_error(DEDISP_MEM_COPY_FAILED);
            }
        }
        else if( !friendly_in_stride ) {
            // Device pointers with unfriendly stride
            if( !copy_device_to_device_2d((dedisp_byte*)d_in,
                                          in_buf_stride_words * BYTES_PER_WORD,
                                          in + gulp_samp_idx*in_stride,
                                          in_stride,
                                          nchan_words * BYTES_PER_WORD,
                                          nsamps_gulp) ) {
                throw_error(DEDISP_MEM_COPY_FAILED);
            }
        }
#ifdef DEDISP_BENCHMARK
        hipDeviceSynchronize();
        copy_to_timer->Pause();
        transpose_timer->Start();
#endif
        // Transpose the words in the input
        transpose(d_in,
                  nchan_words, nsamps_gulp,
                  in_buf_stride_words, nsamps_padded_gulp,
                  d_transposed);
#ifdef DEDISP_BENCHMARK
        hipDeviceSynchronize();
        transpose_timer->Pause();

        kernel_timer->Start();
#endif

        // Unpack the transposed data
        unpack(d_transposed, nsamps_padded_gulp, nchan_words,
               d_unpacked,
               in_nbits, unpacked_in_nbits);

        // Perform direct dedispersion without scrunching
        if( !dedisperse(//d_transposed,
                        d_unpacked,
                        nsamps_padded_gulp,
                        nsamps_computed_gulp,
                        unpacked_in_nbits, //in_nbits,
                        m_nchans,
                        1,
                        thrust::raw_pointer_cast(&d_dm_list[first_dm_idx]),
                        dm_count,
                        1,
                        d_out,
                        out_stride_gulp_samples,
                        out_nbits,
                        1, 0, 0, 0, 0) ) {
            throw_error(DEDISP_INTERNAL_GPU_ERROR);
        }

#ifdef DEDISP_BENCHMARK
        hipDeviceSynchronize();
        kernel_timer->Pause();
#endif
        // Copy output back to host memory if necessary
        if( using_host_memory ) {
            dedisp_size gulp_samp_byte_idx = gulp_samp_idx * out_bytes_per_sample;
            dedisp_size nsamp_bytes_computed_gulp = nsamps_computed_gulp * out_bytes_per_sample;
#ifdef DEDISP_BENCHMARK
            copy_from_timer->Start();
#endif
            copy_device_to_host_2d(out + gulp_samp_byte_idx,  // dst
                                   out_stride,                // dst stride
                                   d_out,                     // src
                                   out_stride_gulp_bytes,     // src stride
                                   nsamp_bytes_computed_gulp, // width bytes
                                   dm_count);                 // height
#ifdef DEDISP_BENCHMARK
            hipDeviceSynchronize();
            copy_from_timer->Pause();
#endif
        }

    } // End of gulp loop

#ifdef DEDISP_BENCHMARK
    cout << "Copy to time:   " << copy_to_timer->ToString() << endl;
    cout << "Copy from time: " << copy_from_timer->ToString() << endl;
    cout << "Transpose time: " << transpose_timer->ToString() << endl;
    cout << "Kernel time:    " << kernel_timer->ToString() << endl;
    auto total_time = copy_to_timer->Milliseconds() +
                      copy_from_timer->Milliseconds() +
                      transpose_timer->Milliseconds() +
                      kernel_timer->Milliseconds();
    cout << "Total time:     " << Stopwatch::ToString(total_time) << endl;

    // Append the timing results to a log file
    std::ofstream perf_file("perf.log", std::ios::app);
    perf_file << copy_to_timer->ToString() << "\t"
              << copy_from_timer->ToString() << "\t"
              << transpose_timer->ToString() << "\t"
              << kernel_timer->ToString() << "\t"
              << Stopwatch::ToString(total_time) << endl;
    perf_file.close();
#endif

    if( !(flags & DEDISP_ASYNC) ) {
        hipStreamSynchronize(stream);
    }
}

void DedispPlan::sync()
{
    if( hipDeviceSynchronize() != hipSuccess )
        throw_error(DEDISP_PRIOR_GPU_ERROR);
}

} // end namespace dedisp