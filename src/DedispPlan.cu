#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "DedispPlan.hpp"
#include "transpose/transpose.hpp"
#include "gpu_memory.hpp"

#include "dedisp_defines.h"
#include "dedisp_error.hpp"
#include "dedisp_kernels.hpp"
#include "dedisperse/dedisperse.h"
#include "unpack/unpack.h"
#include "common/cuda/CU.h"

#if defined(DEDISP_BENCHMARK)
#include <iostream>
#include <fstream>
#include <memory>
using std::cout;
using std::endl;
#include "external/Stopwatch.h"
#endif

namespace dedisp
{

// Constructor
DedispPlan::DedispPlan(size_type  nchans,
                       float_type dt,
                       float_type f0,
                       float_type df)
{
    if( hipGetLastError() != hipSuccess ) {
        throw_error(DEDISP_PRIOR_GPU_ERROR);
    }

    int device_idx;
    hipGetDevice(&device_idx);

    // Check for parameter errors
    if( nchans > DEDISP_MAX_NCHANS ) {
        throw_error(DEDISP_NCHANS_EXCEEDS_LIMIT);
    }

    // Force the df parameter to be negative such that
    //   freq[chan] = f0 + chan * df.
    df = -abs(df);

    m_dm_count      = 0;
    m_nchans        = nchans;
    m_gulp_size     = DEDISP_DEFAULT_GULP_SIZE;
    m_max_delay     = 0;
    m_dt            = dt;
    m_f0            = f0;
    m_df            = df;

    // Generate delay table and copy to device memory
    // Note: The DM factor is left out and applied during dedispersion
    h_delay_table.resize(nchans);
    kernel::generate_delay_table(h_delay_table.data(), nchans, dt, f0, df);
    d_delay_table.resize(nchans * sizeof(dedisp_float));
    htodstream.memcpyHtoDAsync(d_delay_table, h_delay_table.data(), d_delay_table.size());

    // Initialise the killmask
    h_killmask.resize(nchans, (dedisp_bool)true);
    d_killmask.resize(nchans * sizeof(dedisp_bool));
    set_killmask((dedisp_bool*)0);

    htodstream.synchronize();
}

// Destructor
DedispPlan::~DedispPlan() {
}

// Private helper functions
template<typename T>
T min(T a, T b) { return a<b ? a : b; }
unsigned long div_round_up(unsigned long a, unsigned long b) {
    return (a-1) / b + 1;
}

// Public interface
void DedispPlan::set_device(int device_idx) {
    if( hipGetLastError() != hipSuccess ) {
        throw_error(DEDISP_PRIOR_GPU_ERROR);
    }

    hipError_t error = hipSetDevice(device_idx);
    // Note: hipErrorInvalidValue isn't a documented return value, but
    //         it still gets returned :/
    if( hipErrorInvalidDevice == error ||
        hipErrorInvalidValue == error )
        throw_error(DEDISP_INVALID_DEVICE_INDEX);
    else if( hipErrorSetOnActiveProcess == error )
        throw_error(DEDISP_DEVICE_ALREADY_SET);
    else if( hipSuccess != error )
        throw_error(DEDISP_UNKNOWN_ERROR);
}

void DedispPlan::set_gulp_size(size_type gulp_size) {
    m_gulp_size = gulp_size;
}

void DedispPlan::set_killmask(const bool_type* killmask)
{
    if( hipGetLastError() != hipSuccess ) {
        throw_error(DEDISP_PRIOR_GPU_ERROR);
    }
    if( 0 != killmask ) {
        // Copy killmask to plan (both host and device)
        h_killmask.assign(killmask, killmask + m_nchans);
        htodstream.memcpyHtoDAsync(d_killmask, h_killmask.data(), d_killmask.size());
    }
    else {
        // Set the killmask to all true
        std::fill(h_killmask.begin(), h_killmask.end(), (dedisp_bool)true);
        htodstream.memcpyHtoDAsync(d_killmask, h_killmask.data(), d_killmask.size());
    }
}

void DedispPlan::set_dm_list(const float_type* dm_list,
                             size_type         count)
{
    if( !dm_list ) {
        throw_error(DEDISP_INVALID_POINTER);
    }
    if( hipGetLastError() != hipSuccess ) {
        throw_error(DEDISP_PRIOR_GPU_ERROR);
    }

    m_dm_count = count;
    h_dm_list.assign(dm_list, dm_list+count);

    // Copy to the device
    d_dm_list.resize(m_dm_count * sizeof(dedisp_float));
    htodstream.memcpyHtoDAsync(d_dm_list, h_dm_list.data(), d_dm_list.size());
    htodstream.synchronize();

    // Calculate the maximum delay and store it in the plan
    m_max_delay = dedisp_size(h_dm_list[m_dm_count-1] *
                              h_delay_table[m_nchans-1] + 0.5);
}

void DedispPlan::generate_dm_list(float_type dm_start,
                                  float_type dm_end,
                                  float_type ti,
                                  float_type tol)
{
    if( hipGetLastError() != hipSuccess ) {
        throw_error(DEDISP_PRIOR_GPU_ERROR);
    }

    // Generate the DM list (on the host)
    h_dm_list.clear();
    kernel::generate_dm_list(
        h_dm_list,
        dm_start, dm_end,
        m_dt, ti, m_f0, m_df,
        m_nchans, tol);
    m_dm_count = h_dm_list.size();

    // Allocate device memory for the DM list
    d_dm_list.resize(m_dm_count * sizeof(dedisp_float));
    htodstream.memcpyHtoDAsync(d_dm_list, h_dm_list.data(), d_dm_list.size());
    htodstream.synchronize();

    // Calculate the maximum delay and store it in the plan
    m_max_delay = dedisp_size(h_dm_list[m_dm_count-1] *
                              h_delay_table[m_nchans-1] + 0.5);
}

void DedispPlan::execute(size_type        nsamps,
                         const byte_type* in,
                         size_type        in_nbits,
                         byte_type*       out,
                         size_type        out_nbits,
                         unsigned         flags)
{
    enum {
        BITS_PER_BYTE = 8
    };

    // Note: The default out_stride is nsamps - m_max_delay
    dedisp_size out_bytes_per_sample =
        out_nbits / (sizeof(dedisp_byte) * BITS_PER_BYTE);

    // Note: Must be careful with integer division
    dedisp_size in_stride =
        m_nchans * in_nbits / (sizeof(dedisp_byte) * BITS_PER_BYTE);
    dedisp_size out_stride = (nsamps - m_max_delay) * out_bytes_per_sample;

    execute_adv(
        nsamps,
        in, in_nbits, in_stride,
        out, out_nbits, out_stride,
        flags);
}

void DedispPlan::execute_adv(size_type        nsamps,
                             const byte_type* in,
                             size_type        in_nbits,
                             size_type        in_stride,
                             byte_type*       out,
                             size_type        out_nbits,
                             size_type        out_stride,
                             unsigned         flags)
{
    dedisp_size first_dm_idx = 0;
    dedisp_size dm_count = m_dm_count;

    execute_guru(
        nsamps,
        in, in_nbits, in_stride,
        out, out_nbits, out_stride,
        first_dm_idx, dm_count,
        flags);
}

void DedispPlan::execute_guru(size_type        nsamps,
                              const byte_type* in,
                              size_type        in_nbits,
                              size_type        in_stride,
                              byte_type*       out,
                              size_type        out_nbits,
                              size_type        out_stride,
                              size_type        first_dm_idx,
                              size_type        dm_count,
                              unsigned         flags)
{
    if( hipGetLastError() != hipSuccess ) {
        throw_error(DEDISP_PRIOR_GPU_ERROR);
    }

    enum {
        BITS_PER_BYTE  = 8,
        BYTES_PER_WORD = sizeof(dedisp_word) / sizeof(dedisp_byte)
    };

    dedisp_size out_bytes_per_sample = out_nbits / (sizeof(dedisp_byte) *
                                                    BITS_PER_BYTE);

    if( 0 == in || 0 == out ) {
        throw_error(DEDISP_INVALID_POINTER);
    }
    // Note: Must be careful with integer division
    if( in_stride < m_nchans*in_nbits/(sizeof(dedisp_byte)*BITS_PER_BYTE) ||
        out_stride < (nsamps - m_max_delay)*out_bytes_per_sample ) {
        throw_error(DEDISP_INVALID_STRIDE);
    }
    if( 0 == m_dm_count ) {
        throw_error(DEDISP_NO_DM_LIST_SET);
    }
    if( nsamps < m_max_delay ) {
        throw_error(DEDISP_TOO_FEW_NSAMPS);
    }

    // Check for valid synchronisation flags
    if( flags & DEDISP_ASYNC && flags & DEDISP_WAIT ) {
        throw_error(DEDISP_INVALID_FLAG_COMBINATION);
    }

    // Check for valid nbits values
    if( in_nbits  != 1 &&
        in_nbits  != 2 &&
        in_nbits  != 4 &&
        in_nbits  != 8 &&
        in_nbits  != 16 &&
        in_nbits  != 32 ) {
        throw_error(DEDISP_UNSUPPORTED_IN_NBITS);
    }
    if( out_nbits != 8 &&
        out_nbits != 16 &&
        out_nbits != 32 ) {
        throw_error(DEDISP_UNSUPPORTED_OUT_NBITS);
    }

    // Copy the lookup tables to constant memory on the device
    copy_delay_table(d_delay_table,
                     m_nchans * sizeof(dedisp_float),
                     0, 0);
    copy_killmask(d_killmask,
                  m_nchans * sizeof(dedisp_bool),
                  0, 0);

    // Compute the problem decomposition
    dedisp_size nsamps_computed = nsamps - m_max_delay;

    // Specify the maximum gulp size
    dedisp_size nsamps_computed_gulp_max = min(m_gulp_size, nsamps_computed);

    // Just to be sure
    // TODO: This seems quite wrong. Why was it here?
    /*
    if( nsamps_computed_gulp_max < m_max_delay ) {
        throw_error(DEDISP_TOO_FEW_NSAMPS);
    }
    */

    // Compute derived counts for maximum gulp size [dedisp_word == 4 bytes]
    dedisp_size nsamps_gulp_max = nsamps_computed_gulp_max + m_max_delay;
    dedisp_size chans_per_word  = sizeof(dedisp_word)*BITS_PER_BYTE / in_nbits;
    dedisp_size nchan_words     = m_nchans / chans_per_word;

    // We use words for processing but allow arbitrary byte strides, which are
    //   not necessarily friendly.
    //bool friendly_in_stride = (0 == in_stride % BYTES_PER_WORD);

    // Note: If desired, this could be rounded up, e.g., to a power of 2
    dedisp_size in_buf_stride_words      = nchan_words;
    dedisp_size in_count_gulp_max        = nsamps_gulp_max * in_buf_stride_words;

    dedisp_size nsamps_padded_gulp_max   = div_round_up(nsamps_computed_gulp_max,
                                                        DEDISP_SAMPS_PER_THREAD)
        * DEDISP_SAMPS_PER_THREAD + m_max_delay;
    dedisp_size in_count_padded_gulp_max =
        nsamps_padded_gulp_max * in_buf_stride_words;

    // TODO: Make this a parameter?
    dedisp_size min_in_nbits = 0;
    dedisp_size unpacked_in_nbits = max((int)in_nbits, (int)min_in_nbits);
    dedisp_size unpacked_chans_per_word =
        sizeof(dedisp_word)*BITS_PER_BYTE / unpacked_in_nbits;
    dedisp_size unpacked_nchan_words = m_nchans / unpacked_chans_per_word;
    dedisp_size unpacked_buf_stride_words = unpacked_nchan_words;
    dedisp_size unpacked_count_padded_gulp_max =
        nsamps_padded_gulp_max * unpacked_buf_stride_words;

    dedisp_size out_stride_gulp_samples  = nsamps_computed_gulp_max;
    dedisp_size out_stride_gulp_bytes    =
        out_stride_gulp_samples * out_bytes_per_sample;
    dedisp_size out_count_gulp_max       = out_stride_gulp_bytes * dm_count;

    // Organise device memory pointers
    cu::DeviceMemory d_in(in_count_gulp_max * sizeof(dedisp_word));
    cu::DeviceMemory d_transposed(in_count_padded_gulp_max * sizeof(dedisp_word));
    cu::DeviceMemory d_unpacked(unpacked_count_padded_gulp_max * sizeof(dedisp_word));
    cu::DeviceMemory d_out(out_count_gulp_max * sizeof(dedisp_word));

#ifdef DEDISP_BENCHMARK
    std::unique_ptr<Stopwatch> copy_to_timer(Stopwatch::create());
    std::unique_ptr<Stopwatch> copy_from_timer(Stopwatch::create());
    std::unique_ptr<Stopwatch> transpose_timer(Stopwatch::create());
    std::unique_ptr<Stopwatch> kernel_timer(Stopwatch::create());
#endif

    // Gulp loop
    for( dedisp_size gulp_samp_idx=0;
         gulp_samp_idx<nsamps_computed;
         gulp_samp_idx+=nsamps_computed_gulp_max ) {

        dedisp_size nsamps_computed_gulp = min(nsamps_computed_gulp_max,
                                               nsamps_computed-gulp_samp_idx);
        dedisp_size nsamps_gulp          = nsamps_computed_gulp + m_max_delay;
        dedisp_size nsamps_padded_gulp   = div_round_up(nsamps_computed_gulp,
                                                        DEDISP_SAMPS_PER_THREAD)
            * DEDISP_SAMPS_PER_THREAD + m_max_delay;

#ifdef DEDISP_BENCHMARK
        copy_to_timer->Start();
#endif
        // Copy the input data from host to device
        if( !copy_host_to_device_2d((dedisp_byte*)d_in,
                                    in_buf_stride_words * BYTES_PER_WORD,
                                    in + gulp_samp_idx*in_stride,
                                    in_stride,
                                    nchan_words * BYTES_PER_WORD,
                                    nsamps_gulp) ) {
            throw_error(DEDISP_MEM_COPY_FAILED);
        }
#ifdef DEDISP_BENCHMARK
        hipDeviceSynchronize();
        copy_to_timer->Pause();
        transpose_timer->Start();
#endif
        // Transpose the words in the input
        transpose((dedisp_word *) d_in,
                  nchan_words, nsamps_gulp,
                  in_buf_stride_words, nsamps_padded_gulp,
                  (dedisp_word *) d_transposed);
#ifdef DEDISP_BENCHMARK
        hipDeviceSynchronize();
        transpose_timer->Pause();

        kernel_timer->Start();
#endif

        // Unpack the transposed data
        unpack(d_transposed, nsamps_padded_gulp, nchan_words,
               d_unpacked,
               in_nbits, unpacked_in_nbits);

        // Perform direct dedispersion without scrunching
        if( !dedisperse(//d_transposed,
                        d_unpacked,
                        nsamps_padded_gulp,
                        nsamps_computed_gulp,
                        unpacked_in_nbits, //in_nbits,
                        m_nchans,
                        1,
                        d_dm_list,
                        dm_count,
                        1,
                        d_out,
                        out_stride_gulp_samples,
                        out_nbits,
                        1, 0, 0, 0, 0) ) {
            throw_error(DEDISP_INTERNAL_GPU_ERROR);
        }

#ifdef DEDISP_BENCHMARK
        hipDeviceSynchronize();
        kernel_timer->Pause();
#endif
        // Copy output back to host memory
        dedisp_size gulp_samp_byte_idx = gulp_samp_idx * out_bytes_per_sample;
        dedisp_size nsamp_bytes_computed_gulp = nsamps_computed_gulp * out_bytes_per_sample;
#ifdef DEDISP_BENCHMARK
        copy_from_timer->Start();
#endif
        copy_device_to_host_2d(out + gulp_samp_byte_idx,  // dst
                                out_stride,                // dst stride
                                (byte_type *) d_out,       // src
                                out_stride_gulp_bytes,     // src stride
                                nsamp_bytes_computed_gulp, // width bytes
                                dm_count);                 // height
#ifdef DEDISP_BENCHMARK
        hipDeviceSynchronize();
        copy_from_timer->Pause();
#endif

    } // End of gulp loop

#ifdef DEDISP_BENCHMARK
    cout << "Copy to time:   " << copy_to_timer->ToString() << endl;
    cout << "Copy from time: " << copy_from_timer->ToString() << endl;
    cout << "Transpose time: " << transpose_timer->ToString() << endl;
    cout << "Kernel time:    " << kernel_timer->ToString() << endl;
    auto total_time = copy_to_timer->Milliseconds() +
                      copy_from_timer->Milliseconds() +
                      transpose_timer->Milliseconds() +
                      kernel_timer->Milliseconds();
    cout << "Total time:     " << Stopwatch::ToString(total_time) << endl;

    // Append the timing results to a log file
    std::ofstream perf_file("perf.log", std::ios::app);
    perf_file << copy_to_timer->ToString() << "\t"
              << copy_from_timer->ToString() << "\t"
              << transpose_timer->ToString() << "\t"
              << kernel_timer->ToString() << "\t"
              << Stopwatch::ToString(total_time) << endl;
    perf_file.close();
#endif
}

void DedispPlan::sync()
{
    if( hipDeviceSynchronize() != hipSuccess )
        throw_error(DEDISP_PRIOR_GPU_ERROR);
}

} // end namespace dedisp